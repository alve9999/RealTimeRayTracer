﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "common.h"
#include <cstdlib>
#include "SDL.h"
#include <chrono>
#include "pixel_buffer.h"
#include "SDL_container.h"
#include "g-truc-glm-bf71a83/glm/glm.hpp"
#include "Ray.h"

void trace_ray(Ray ray) {

}

int main(int argc, char* argv[]) {
    //sets random number generator
    srand(std::chrono::system_clock::to_time_t(std::chrono::system_clock::now()));

    //init SDL
    SDL_container window;

    //initialise the pixel buffer into unified memory
    pixel_buffer image(screen_height, screen_width);

    timer_start;
    timer_end;
    window.render_pixel_buffer(image);



    return 0;
}


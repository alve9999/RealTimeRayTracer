#include "hip/hip_runtime.h"
﻿#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "common.h"
#include <cstdlib>
#include "SDL.h"
#include <chrono>
#include "pixel_buffer.h"
#include "SDL_container.h"
#include "g-truc-glm-bf71a83/glm/glm.hpp"
#include "ray.h"
#include "camera.h"
#include "scene.h"

void render(glm::vec3* image, camera* main_camera, scene* main_scene,float sampels) {
    for (int i = 0; i < screen_height * screen_width; i++) {
        ray tracing_ray = main_camera->get_ray(((i % screen_width - screen_width/2) * 2 + ((RAND) - 0.5)) / screen_width, (2 * (std::floor((float)i / screen_height) - screen_height/2) + (RAND - 0.5)) / screen_height);
        glm::vec3 ray_colour = main_scene->trace_ray(tracing_ray);
        glm::vec3 pixel_colour = image[i / screen_width * screen_width + i % screen_width];
        pixel_colour = (pixel_colour * (sampels - 1.0f) / sampels + ray_colour / sampels);
    }
}



int main(int argc, char* argv[]) {
    //sets random number generator
    srand(std::chrono::system_clock::to_time_t(std::chrono::system_clock::now()));

    //init SDL
    SDL_container window;


    glm::vec3* image = (glm::vec3*)malloc(screen_height*screen_width*sizeof(glm::vec3));
    for (int i = 0; i < screen_height * screen_width; i++) {
        image[i / screen_width * screen_width + i % screen_width] = glm::vec3(0, 0, 0);
    }
    float sampels = 0;

    //init scene
    scene* main_scene;
    main_scene = new scene();

    //init camera
    camera* main_camera;
    main_camera = new camera(glm::vec3(0,0,0),PI*2/3,glm::vec3(1, 1, 0));


    while (1) {
        render(image, main_camera, main_scene,sampels);
        window.render_pixel_buffer(image);
        sampels += 1;
        std::cout << sampels;
    }



    return 0;
}

